#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_fp16.h>
#include <functional>

#include "cuda_attn.hpp"
#include "profile_utils.cuh"

int main()
{   
    print_device_info();
    unsigned int batch_size = 8U;
    unsigned int num_heads = 16U;
    unsigned int seq_len = 256U;
    unsigned int head_dim = 64U; // just implement for 32 and 64

    unsigned int num_warmups = 1U;
    unsigned int num_repeats = 1U;

    __half const abs_tol{__float2half(5.0e-2f)};
    double const rel_tol{1.0e-1f};

    // print attention settings
    std::cout << "batch size = " << batch_size << std::endl;
    std::cout << "sequence length = " << seq_len << std::endl;
    std::cout << "number of heads = " << num_heads << std::endl;
    std::cout << "dimension = " << head_dim << std::endl;

    const std::vector <
        std::pair<
            std::string,
            std::function<
                void(
                    const __half*,
                    const __half*,
                    const __half*,
                    __half*,
                    unsigned int, unsigned int, unsigned int, unsigned int,
                    hipStream_t stream
                )
            >
        >
    > attention_launchers = {
        {"cuda core flash attention 01", launch_flash_attention_01<__half>},
        {"cuda core flash attention 02", launch_flash_attention_02<__half>},
        {"cute flash attention 02", mha_fwd<__half>},
    };

    for (const auto& [name, attention_launcher] : attention_launchers) {
        std::cout << "-------------------------------------------------" << std::endl;
        std::cout << "implementation: " << name << std::endl;
        std::pair<float, float> results = profile_attention<__half>(
            batch_size, num_heads, seq_len, head_dim,
            attention_launcher,
            num_warmups, num_repeats,
            abs_tol, rel_tol
        );
        std::cout << std::endl;
    }

    return 0;
}