#include <iostream>

#include "profile_utils.cuh"
#include "cuda_utils.hpp"

void print_device_info()
{   
    // get device
    int device_id = 0;
    hipGetDevice(&device_id);
    
    // get device properties
    hipDeviceProp_t device_prop;
    hipGetDeviceProperties(&device_prop, device_id);

    // print device name
    std::cout << "Device name: " << device_prop.name << std::endl;

    // print global memory size in GB
    float const global_memory_size{static_cast<float>(device_prop.totalGlobalMem / (1 << 30))}; // 1 << 30 bytes = 1 GiB ~ 1 GB 
    std::cout << "Global memory size: " << global_memory_size << " GB" << std::endl;

    // print peak memory bandwidth
    float const peak_bandwidth{static_cast<float>((2.0f * device_prop.memoryClockRate * device_prop.memoryBusWidth / 8) / 1.0e6)}; // 1.0e6 from kHz to Ghz
    std::cout << "Peak memory bandwidth: " << peak_bandwidth << " GB/s" << std::endl;

    std::cout << std::endl;
}

float measure_performance(
    std::function<void(hipStream_t)> launcher, 
    hipStream_t stream, 
    const int num_warmups, 
    const int num_repeats)
{
    // Event initialization
    float milliseconds = 0.0f;
    hipEvent_t start, stop;
    CHECK_CUDA_ERROR(hipEventCreate(&start));
    CHECK_CUDA_ERROR(hipEventCreate(&stop));

    // warmup
    for (unsigned int i = 0; i < num_warmups; i++)
    {
        launcher(stream);
    }

    // synchronize
    hipStreamSynchronize(stream);

    // benchmark
    CHECK_CUDA_ERROR(hipEventRecord(start, stream));
    for (unsigned int i = 0; i < num_repeats; i++)
    {
        launcher(stream);
    }
    CHECK_CUDA_ERROR(hipEventRecord(stop, stream));

    // synchronize
    CHECK_CUDA_ERROR(hipEventSynchronize(stop));
    CHECK_LAST_CUDA_ERROR();

    // get result and destroy events
    CHECK_CUDA_ERROR(hipEventElapsedTime(&milliseconds, start, stop));
    CHECK_CUDA_ERROR(hipEventDestroy(start));
    CHECK_CUDA_ERROR(hipEventDestroy(stop));

    return milliseconds / static_cast<float>(num_repeats);
}

