#include <iostream>
#include <hip/hip_runtime.h>

#include "cuda_utils.hpp"

void check_cuda_error(hipError_t val, const char* const func, const char* const file, const int line)
{
    if (val != hipSuccess)
    {
        std::cerr << "CUDA runtime error at: " << file << ":" << line << std::endl;
        std::cerr << "CUDA error message: " << hipGetErrorString(val) << " " << func << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

void check_last_cuda_error(const char* const file, const int line)
{
    const hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        std::cerr << "CUDA runtime error at: " << file << ":" << line << std::endl;
        std::cerr << "CUDA error message: " << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}